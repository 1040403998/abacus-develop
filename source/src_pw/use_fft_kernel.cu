#include "hip/hip_runtime.h"
#include "use_fft_kernel.h"
#include "global.h"
#include "hipfft/hipfft.h"

__global__ void kernel_set(int size, CUFFT_COMPLEX *dst, const CUFFT_COMPLEX *src, const int *index_list)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int p = index_list[idx];
    if(idx < size)
    {
        dst[p].x = src[idx].x;
        dst[p].y = src[idx].y;
    }
}

__global__ void kernel_roundtrip(int size, CUFFT_COMPLEX *dst, const double *src)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        dst[idx].x *= src[idx];
        dst[idx].y *= src[idx];
    }
}

__global__ void kernel_normalization(int size, CUFFT_COMPLEX *data, double norm)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        data[idx].x /= norm;
        data[idx].y /= norm;
    }
}

void UfftRoundtripKernel(const CUFFT_COMPLEX *psi, const double *vr, const int *fft_index, CUFFT_COMPLEX *psic)
{
    // cout<<"fft dim before ufft: "<<GlobalC::pw.nx<<" "<<GlobalC::pw.ny<<" "<<GlobalC::pw.nz<<endl;
    // cout<<"rounftrip on GPU!"<<endl;

    // cout<<"before set"<<endl;
    // complex<double> *psic_inside = new complex<double>[15];
    // hipMemcpy(psic_inside, &psic[6000], 15*sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToHost);
    // for(int i=0;i<15;i++)
    // {
    //     cout<<psic_inside[i].real()<<" "<<psic_inside[i].imag()<<endl;
    // }
    // // delete [] psic_inside;
    // cout<<"========"<<endl;

    // (1) set value
    int thread = 512;
    int block = GlobalC::wf.npw / thread + 1;
    kernel_set<<<block, thread>>>(GlobalC::wf.npw, psic, psi, fft_index);

    // cout<<"before fft1"<<endl;
    // complex<double> *psic_inside = new complex<double>[15];
    // hipMemcpy(psic_inside, &psic[6000], 15*sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToHost);
    // for(int i=0;i<15;i++)
    // {
    //     cout<<psic_inside[i].real()<<" "<<psic_inside[i].imag()<<endl;
    // }
    // // delete [] psic_inside;
    // cout<<"========"<<endl;


    // for(int ig=0;ig<wf.npw;ig++)
    // {
    //     psic[fft_index[ig]] = psi[ig];
    // }


    // hipfftHandle cufftplan_gpu;
    // hipfftPlan3d(&cufftplan_gpu, GlobalC::pw.nx, GlobalC::pw.ny, GlobalC::pw.nz, HIPFFT_Z2Z);
    // hipfftExecZ2Z(cufftplan_gpu, psic, psic, HIPFFT_BACKWARD);
    // hipfftDestroy(cufftplan_gpu);

    complex<double> *psic_cpu = new complex<double>[GlobalC::pw.nrxx];
    hipMemcpy(psic_cpu, psic, GlobalC::pw.nrxx*sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToHost);
    GlobalC::pw.FFT_wfc.FFT3D( psic_cpu, 1);

    hipMemcpy(psic, psic_cpu, GlobalC::pw.nrxx*sizeof(CUFFT_COMPLEX), hipMemcpyHostToDevice);

    // cout<<"after fft1"<<endl;
    // psic_inside = new complex<double>[15];
    // hipMemcpy(psic_inside, &psic[6000], 15*sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToHost);
    // for(int i=0;i<15;i++)
    // {
    //     cout<<psic_inside[i].real()<<" "<<psic_inside[i].imag()<<endl;
    // }
    // // delete [] psic_inside;
    // cout<<"========"<<endl;

    // double *vr_cpu = new double[15];
    // hipMemcpy(vr_cpu, &vr[6000], sizeof(double)*15, hipMemcpyDeviceToHost);
    // cout<<"vr ERROR:"<<endl;
    // for(int i=0;i<15;i++){
    //     cout<<vr_cpu[i]<<endl;
    // }
    // cout<<"==========="<<endl;
 
    int block2 = GlobalC::pw.nrxx / thread + 1;
    kernel_roundtrip<<<block2, thread>>>(GlobalC::pw.nrxx, psic, vr);

    // cout<<"before fft2"<<endl;
    // psic_inside = new complex<double>[15];
    // hipMemcpy(psic_inside, &psic[6000], 15*sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToHost);
    // for(int i=0;i<15;i++)
    // {
    //     cout<<psic_inside[i].real()<<" "<<psic_inside[i].imag()<<endl;
    // }
    // delete [] psic_inside;
    // cout<<"========"<<endl;

    // (3) fft back to G space
    // hipfftHandle cufftplan_gpu2;
    // hipfftPlan3d(&cufftplan_gpu2, GlobalC::pw.nx, GlobalC::pw.ny, GlobalC::pw.nz, HIPFFT_Z2Z);
    // hipfftExecZ2Z(cufftplan_gpu2, psic, psic, HIPFFT_FORWARD);

    // hipfftDestroy(cufftplan_gpu2);

    // cout<<"before normalization"<<endl;
    // complex<double> *tmp2 = new complex<double>[15];
    // hipMemcpy(tmp2, &psic[6000], 15*sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToHost);
    // for(int i=0;i<15;i++)
    // {
    //     cout<<tmp2[i].real()<<" "<<tmp2[i].imag()<<endl;
    // }
    // delete [] tmp2;

    // int block3 = GlobalC::pw.nrxx / thread + 1;
    // kernel_normalization<<<block3, thread>>>(GlobalC::pw.nrxx, psic, (double)(GlobalC::pw.nrxx));
    
    // complex<double> *psic_cpu = new complex<double>[GlobalC::pw.nrxx];
    psic_cpu = new complex<double>[GlobalC::pw.nrxx];
    hipMemcpy(psic_cpu, psic, GlobalC::pw.nrxx*sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToHost);
    GlobalC::pw.FFT_wfc.FFT3D( psic_cpu, -1);
    hipMemcpy(psic, psic_cpu, GlobalC::pw.nrxx*sizeof(CUFFT_COMPLEX), hipMemcpyHostToDevice);

    // cout<<"after 2nd ufft SUCCESS"<<endl;
    // complex<double> *tmp1 = new complex<double>[15];
    // hipMemcpy(tmp1, &psic[6000], 15*sizeof(CUFFT_COMPLEX), hipMemcpyDeviceToHost);
    // for(int i=0;i<15;i++)
    // {
    //     cout<<tmp1[i].real()<<" "<<tmp1[i].imag()<<endl;
    // }
    // delete [] tmp1;

    // cout<<"rounftrip end"<<endl;

    // cout<<"fft dim: "<<GlobalC::pw.nx<<" "<<GlobalC::pw.ny<<" "<<GlobalC::pw.nz<<endl;
    return;
}